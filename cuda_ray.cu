#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define CUDA 0
#define OPENMP 1
#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
    float    r,b,g;
    float    radius;
    float    x,y,z;
    __device__ float  hit( float  ox, float  oy, float  *n ) {
        float  dx = ox - x;
        float  dy = oy - y;
        if  (dx*dx + dy*dy < radius*radius) {
            float  dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return  dz + z;
        }
        return  -INF;
    }
};

void ppm_write(unsigned char* bitmap, int  xdim,int  ydim, FILE* fp)
{
  int  i,x,y;
  fprintf(fp,"P3\n");
  fprintf(fp,"%d %d\n",xdim, ydim);
  fprintf(fp,"255\n");
  for  (y=0 ;y<ydim;y++) {
    for  (x=0 ;x<xdim;x++) {
      i=x+y*xdim;
      fprintf(fp,"%d %d %d ",bitmap[4 *i],bitmap[4 *i+1 ],bitmap[4 *i+2 ]);
    }
    fprintf(fp,"\n");
  }
}

__global__ void kernel(Sphere *s, unsigned char *ptr)
{
  int  x = threadIdx.x + blockIdx.x * blockDim.x;
  int  y = threadIdx.y + blockIdx.y * blockDim.y;
  int  offset = x + y * blockDim.x * gridDim.x;
  float  ox = (x - DIM/2 );
  float  oy = (y - DIM/2 );

  float  r=0 , g=0 , b=0 ;
  float    maxz = -INF;
  for (int  i=0 ; i<SPHERES; i++) {
    float    n;
    float    t = s[i].hit( ox, oy, &n );
    if  (t > maxz) {
      float  fscale = n;
      r = s[i].r * fscale;
      g = s[i].g * fscale;
      b = s[i].b * fscale;
      maxz = t;
    } 
  }
  ptr[offset*4  + 0 ] = (int )(r * 255 );
  ptr[offset*4  + 1 ] = (int )(g * 255 );
  ptr[offset*4  + 2 ] = (int )(b * 255 );
  ptr[offset*4  + 3 ] = 255 ;
}

int  main(void)
{
  int  x,y;
  hipEvent_t start, end;
  float  execTime;

  hipEventCreate(&start);
  hipEventCreate(&end);

  unsigned char* bitmap;
  unsigned char* d_bitmap;
  Sphere *d_s;

  srand(time(NULL));
  FILE* fp = fopen("result.ppm","w");

  hipEventRecord(start, 0 ); // timer start

  Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
  
  for  (int  i=0 ; i<SPHERES; i++) {
    temp_s[i].r = rnd( 1.0 f );
    temp_s[i].g = rnd( 1.0 f );
    temp_s[i].b = rnd( 1.0 f );
    temp_s[i].x = rnd( 2000.0 f ) - 1000 ;
    temp_s[i].y = rnd( 2000.0 f ) - 1000 ;
    temp_s[i].z = rnd( 2000.0 f ) - 1000 ;
    temp_s[i].radius = rnd( 200.0 f ) + 40 ;
  }
  
  bitmap=(unsigned char*)malloc(sizeof(unsigned char)*DIM*DIM*4 );

  // memory allocation to use in device memory
  hipMalloc((void**)&d_bitmap, sizeof(unsigned char)*DIM*DIM*4 );
  hipMalloc((void**)&d_s, sizeof(Sphere)*SPHERES);


  // memory copy from host(temp_s) to device(d_s)
  hipMemcpy(d_s, temp_s, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice);
  
  // configures the number of grid and thread 
  dim3 grids(DIM/16 , DIM/16 );
  dim3 threads(16 , 16 );

  // call kernel function
  kernel<<<grids, threads>>>(d_s, d_bitmap);

  // memory copy from device(d_bitmap; result) to host(bitmap)
  hipMemcpy(bitmap, d_bitmap, sizeof(unsigned char)*DIM*DIM*4 , hipMemcpyDeviceToHost);

  hipEventRecord(end, 0 ); // timer stop
  hipEventSynchronize(end);
  hipEventElapsedTime(&execTime, start, end); // caculate execution time
    
  ppm_write(bitmap,DIM,DIM,fp); // writing result in a result file

  printf("CUDA ray tracing: %lf sec\n", execTime * 0.001 );
  printf("[result.ppm] was generated.\n");
  fclose(fp);
  hipEventDestroy(start);
  hipEventDestroy(end);
  free(bitmap);
  free(temp_s);

  return  0 ;
}
